// Matrix Multiplication in gpu with 2D grid of blocks with 1D block shape
// Compile with: nvcc -o test matrix_multiplication_2D_2D.cu -std=c++11


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <chrono>
 
// Multiplies matrices using GPU with 2D grid
__global__ void multiply_matrix_gpu(long *matA, long *matB, long *matC, const int n) {
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y;

    if (ix < n && iy < n) {
        for(int k=0; k<n; k++) {
            matC[iy*n+ix] += matA[iy*n+k] * matB[k*n+ix];
        }
    }
}

// Multiplies matrices in host
void multiply_matrix_host(long *matA, long *matB, long *matC, int n) {
    for(int i = 0; i<n; i++) {
        for(int j=0; j<n; j++) {
            for(int k=0; k<n; k++) {
                matC[i*n+j] += matA[i*n+k] * matB[j+k*n];
            }
        }
    }
}

// Compares two matrices
void checkResult(long *hostRef, long *gpuRef, const int n) {
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < n*n; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("host %ld gpu %ld\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match) printf("Matrix match.\n\n");
    else printf("Matrix does not not match.\n\n");
}
 
int main(int argc, char* argv[]) {
    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    // Size of matrix
    int n = 1000;
    int bytes = n * n * sizeof(long*);

    // Host matrix memory
    long *h_a = (long *)malloc(bytes);
    long *h_b = (long *)malloc(bytes);

    // Results
    long *hostRef = (long *)malloc(bytes);
    long *gpuRef = (long *)malloc(bytes);

    // Initialize matrix on host
    for(int i = 0; i < n*n; i++ ) {
        h_a[i] = i+1;
        h_b[i] = i+1;
    }

    // Initialize matrix with 0s
    memset(hostRef, 0, bytes);
    memset(gpuRef, 0, bytes);

    // Multiply matrix on host
    auto start_cpu = std::chrono::high_resolution_clock::now();
    multiply_matrix_host(h_a, h_b, hostRef, n);
    auto end_cpu =  std::chrono::high_resolution_clock::now();

    // Measure total time in host
    std::chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;
    printf("multiply_matrix_host elapsed %f ms\n", duration_ms.count());

    // Device matrix global memory
    long *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, bytes);
    hipMalloc((void **)&d_b, bytes);
    hipMalloc((void **)&d_c, bytes);

    // Transfer data from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    hipMemset(d_c, 0, bytes);  // Initialize matrix with 0s

    // Kernel execution configuration
    dim3 block(128);
    dim3 grid((n + block.x - 1) / block.x, n);
    printf("grid.x %d grid.y %d block.x %d \n", grid.x, grid.y, block.x);

    // Execute kernel
    start_cpu = std::chrono::high_resolution_clock::now();
    multiply_matrix_gpu<<<grid, block>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();
    end_cpu =  std::chrono::high_resolution_clock::now();

    // Measure total time
    duration_ms = end_cpu - start_cpu;
    printf("multiply_matrix_gpu elapsed %f ms\n", duration_ms.count());

    // Copy result from device to host
    hipMemcpy(gpuRef, d_c, bytes, hipMemcpyDeviceToHost);

    // Check results
    checkResult(hostRef, gpuRef, n);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(hostRef);
    free(gpuRef);
    
    hipDeviceReset();

    return 0;
}
