/* 
 * Matrix Multiplication in gpu with 2D grid of blocks
 * https://imgur.com/DHGl22F
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <chrono>
 
__global__ void multiply_matrix_gpu(long* matA, long* matB, long* matC, const int n) {
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < n && iy < n) {
        for(int k=0; k<n; k++) {
            matC[iy*n+ix] += matA[iy*n+k] * matB[k*n+ix];
        }
    }
}

void multiply_matrix_host(long* input_matrix_a, long* input_matrix_b, long* output_matrix, const int n) {
    for(int i = 0; i<n; i++) {
        for(int j=0; j<n; j++) {
            for(int k=0; k<n; k++) {
                output_matrix[i*n+j] += input_matrix_a[i*n+k] * input_matrix_b[j+k*n];
            }
        }
    }
}

void checkResult(long *hostRef, long *gpuRef, const int n) {
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < n*n; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("host %ld gpu %ld\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match) printf("Matrix match.\n\n");
    else printf("Matrix does not not match.\n\n");
}
 
int main(int argc, char* argv[]) {
    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    // Size of matrix
    int n = 3;
    int bytes = n * n * sizeof(long*);

    // Host matrix memory
    long *h_a = (long *)malloc(bytes);
    long *h_b = (long *)malloc(bytes);

    // Results
    long *hostRef = (long *)malloc(bytes);
    long *gpuRef = (long *)malloc(bytes);

    // Initialize matrix on host
    for(int i = 0; i < n*n; i++ ) {
        h_a[i] = i+1;
        h_b[i] = i+1;
    }

    // Initialize matrix with 0s
    memset(hostRef, 0, bytes);
    memset(gpuRef, 0, bytes);

    // Multiply matrix on host
    auto start_cpu = std::chrono::high_resolution_clock::now();
    multiply_matrix_host(h_a, h_b, hostRef, n);
    auto end_cpu =  std::chrono::high_resolution_clock::now();

    // Measure total time in host
    std::chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;
    printf("multiply_matrix_host elapsed %f ms\n", duration_ms.count());

    // Device matrix global memory
    long *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, bytes);
    hipMalloc((void **)&d_b, bytes);
    hipMalloc((void **)&d_c, bytes);

    // Transfer data from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    hipMemset(d_c, 0, bytes);  // Initialize matrix with 0s

    // Kernel execution configuration
    dim3 block(32, 32);
    dim3 grid((n + block.x - 1) / block.x, (n + block.y - 1) / block.y);
    printf("grid.x %d grid.y %d block.x %d block.y %d\n", grid.x, grid.y, block.x, block.y);

    // Execute kernel
    start_cpu = std::chrono::high_resolution_clock::now();
    multiply_matrix_gpu<<<grid, block>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();
    end_cpu =  std::chrono::high_resolution_clock::now();

    // Measure total time
    duration_ms = end_cpu - start_cpu;
    printf("multiply_matrix_gpu elapsed %f ms\n", duration_ms.count());

    // Copy result from device to host
    hipMemcpy(gpuRef, d_c, bytes, hipMemcpyDeviceToHost);

    // Check results
    checkResult(hostRef, gpuRef, n);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(hostRef);
    free(gpuRef);
    
    hipDeviceReset();

    return 0;
}
